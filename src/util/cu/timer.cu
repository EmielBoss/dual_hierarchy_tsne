/*
 * MIT License
 *
 * Copyright (c) 2021 Mark van de Ruit (Delft University of Technology)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <utility>
#include <hip/hip_runtime.h>
#include "util/cu/timer.cuh"

namespace dh::util {
  CUTimer::CUTimer() : Timer() {
    hipEventCreate((hipEvent_t *) &_startHandle);
    hipEventCreate((hipEvent_t *) &_stopHandle);
  }

  CUTimer::~CUTimer() {
    hipEventDestroy((hipEvent_t) _startHandle);
    hipEventDestroy((hipEvent_t) _stopHandle);
  }

  CUTimer::CUTimer(CUTimer&& other) noexcept {
    swap(*this, other);
  }

  CUTimer& CUTimer::operator=(CUTimer&& other) noexcept {
    swap(*this, other);
    return *this;
  }

  void swap(CUTimer& a, CUTimer& b) noexcept {
    using std::swap;
    swap(a._values, b._values);
    swap(a._iterations, b._iterations);
    swap(a._startHandle, b._startHandle);
    swap(a._stopHandle, b._stopHandle);
  }

  void CUTimer::tick() {
    hipEventRecord((hipEvent_t) _startHandle);
  }

  void CUTimer::tock() {
    hipEventRecord((hipEvent_t) _stopHandle);
  }

  void CUTimer::poll() {
    float fElapsed;
    
    // Query elapsed time (maximum microsecond resolution)
    hipEventSynchronize((hipEvent_t) _stopHandle);
    hipEventElapsedTime(&fElapsed, (hipEvent_t) _startHandle, (hipEvent_t) _stopHandle);
    long long elapsed = static_cast<long long>(1000000.f * fElapsed);

    // Update last, total, average times
    _values(TimerValue::eLast) = std::chrono::nanoseconds(elapsed);
    _values(TimerValue::eTotal) += _values(TimerValue::eLast);
    _values(TimerValue::eAverage) = _values(TimerValue::eAverage)
      + (_values(TimerValue::eLast) - _values(TimerValue::eAverage)) / (++_iterations);
  }
}