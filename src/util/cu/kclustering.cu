/*
 * MIT License
 *
 * Copyright (c) 2021 Mark van de Ruit (Delft University of Technology)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

// #include <iostream>
#include <hip/hip_runtime.h>
#include <faiss/Clustering.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <glad/glad.h>
#include "dh/util/cu/kclustering.cuh"
#include "dh/util/cu/knn.cuh"
// #include "dh/util/cu/error.cuh"
// #include "dh/util/gl/error.hpp"
// #include "dh/util/io.hpp"

namespace dh::util {

  KClustering::KClustering() 
  : _isInit(false), _n(0), _d(0) {
    // ...
  }

  KClustering::KClustering(uint n, uint d, GLuint dataBufferHandle, GLuint indicesBufferHandle)
  : _isInit(false), _n(n), _d(d) {

    _buffers(BufferType::eDatasetIn) = dataBufferHandle;
    _buffers(BufferType::eIndicesIn) = indicesBufferHandle;
    glCreateBuffers(1, &_buffers(BufferType::eDatasetOut));
    glCreateBuffers(1, &_buffers(BufferType::eIndicesOut));
    _isInit = true;
  }

  KClustering::~KClustering() {
    if (_isInit) {
      glDeleteBuffers(_buffers.size(), _buffers.data());
      glDeleteBuffers(_buffersTemp.size(), _buffersTemp.data());
    }
  }

  KClustering::KClustering(KClustering&& other) noexcept {
    swap(*this, other);
  }

  KClustering& KClustering::operator=(KClustering&& other) noexcept {
    swap(*this, other);
    return *this;
  }

  void KClustering::comp(uint nCentroids, bool medoids) {

    // Annoyingly, Faiss' clustering doesn't support GPU buffers, so we have to copy data over
    std::vector<float> dataIn(_n * _d);
    glGetNamedBufferSubData(_buffers(BufferType::eDatasetIn), 0, _n * _d * sizeof(float), dataIn.data());

    // Use a single GPU device. For now, just grab device 0 and pray
    faiss::gpu::StandardGpuResources faissResources;
    faiss::gpu::GpuIndexFlatConfig faissConfig;
    faissConfig.device = 0;
    faissConfig.useFloat16 = false;
    faiss::gpu::GpuIndexFlatL2 faissIndex(&faissResources, _d, faissConfig);

    faiss::ClusteringParameters faissClusteringParams;
    faissClusteringParams.niter = 30;
    faissClusteringParams.verbose = false;
    faissClusteringParams.seed = 0;
    faiss::Clustering kMeans(_d, nCentroids, faissClusteringParams);

    kMeans.train(_n, dataIn.data(), faissIndex);

    if(medoids) {
      glCreateBuffers(_buffersTemp.size(), _buffersTemp.data());
      glNamedBufferStorage(_buffers(BufferType::eIndicesOut), nCentroids * sizeof(uint), nullptr, 0);
      glNamedBufferStorage(_buffersTemp(BufferTempType::eDistances), nCentroids * _d * sizeof(float), nullptr, 0);
      util::KNN knn(_buffers(BufferType::eDatasetIn), _buffersTemp(BufferTempType::eDistances), _buffers(BufferType::eIndicesOut), _n, 1, _d);
      knn.comp(kMeans.centroids.data(), nCentroids);
      
      std::vector<uint> indices(nCentroids);
      glGetNamedBufferSubData(_buffers(BufferType::eIndicesOut), 0, nCentroids * sizeof(uint), indices.data());
      glNamedBufferStorage(_buffers(BufferType::eDatasetOut), nCentroids * _d * sizeof(float), nullptr, GL_DYNAMIC_STORAGE_BIT);
      for (uint i = 0; i < nCentroids; ++i) {
        glCopyNamedBufferSubData(_buffers(BufferType::eDatasetIn), _buffers(BufferType::eDatasetOut), indices[i] * _d * sizeof(float), i * _d * sizeof(float), _d * sizeof(float));
      }

      glDeleteBuffers(_buffersTemp.size(), _buffersTemp.data());
    } else {
      glNamedBufferStorage(_buffers(BufferType::eDatasetOut), nCentroids * _d * sizeof(float), kMeans.centroids.data(), 0);
    }

  }
} // dh::util