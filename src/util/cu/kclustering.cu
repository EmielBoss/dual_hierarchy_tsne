/*
 * MIT License
 *
 * Copyright (c) 2021 Mark van de Ruit (Delft University of Technology)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <faiss/Clustering.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <glad/glad.h>
#include "dh/util/cu/kclustering.cuh"
#include "dh/util/cu/knn.cuh"
#include "dh/util/cu/error.cuh"
#include "dh/util/gl/error.hpp"
#include "dh/util/io.hpp"

namespace dh::util {

  KClustering::KClustering() 
  : _isInit(false), _n(0), _d(0), _dataPtr(nullptr) {
    // ...
  }

  KClustering::KClustering(const float* dataPtr, uint n, uint d)
  : _isInit(false), _n(n), _d(d), _dataPtr(dataPtr) {

    _isInit = true;
  }

  // KClustering::KClustering(GLuint datasetBuffer, uint n, uint d, uint levels)
  // : _isInit(false), _n(n), _d(d), _levels(levels), _dataPtr(nullptr) {
    
  //   // Set up OpenGL-CUDA interoperability
  //   _interopBuffers(BufferType::eDataset) = CUGLInteropBuffer(datasetBuffer, CUGLInteropType::eNone);
  //   // _interopBuffers(BufferType::eCentroids) = CUGLInteropBuffer(centroidsBuffer, CUGLInteropType::eNone);

  //   _isInit = true;
  // }

  KClustering::~KClustering() {
    if (_isInit) {
      glDeleteBuffers(1, &_bufferResults);
    }
  }

  KClustering::KClustering(KClustering&& other) noexcept {
    swap(*this, other);
  }

  KClustering& KClustering::operator=(KClustering&& other) noexcept {
    swap(*this, other);
    return *this;
  }

  void KClustering::comp(uint nCentroids, bool medoids) {
    // Map interop buffers for access on CUDA side
    // _interopBuffers(BufferType::eCentroids).map();

    const float* dataPtr;
    if(_dataPtr) { dataPtr = _dataPtr; }
    else {
      _interopBuffers(BufferType::eDataset).map();
      dataPtr = (float*) _interopBuffers(BufferType::eDataset).cuHandle();
    }

    // Use a single GPU device. For now, just grab device 0 and pray
    faiss::gpu::StandardGpuResources faissResources;
    faiss::gpu::GpuIndexFlatConfig faissConfig;
    faissConfig.device = 0;
    faissConfig.useFloat16 = false;
    faiss::gpu::GpuIndexFlatL2 faissIndex(&faissResources, _d, faissConfig);

    faiss::ClusteringParameters faissClusteringParams;
    faissClusteringParams.niter = 30;
    faissClusteringParams.verbose = false;
    faissClusteringParams.seed = 0;
    faiss::Clustering kMeans(_d, nCentroids, faissClusteringParams);

    kMeans.train(_n, dataPtr, faissIndex);

    glCreateBuffers(1, &_bufferResults);
    if(medoids) {
      glCreateBuffers(_buffersMedoids.size(), _buffersMedoids.data());
      glNamedBufferStorage(_buffersMedoids(BufferMedoidsType::eDataset), _n * _d * sizeof(float), dataPtr, 0);
      glNamedBufferStorage(_buffersMedoids(BufferMedoidsType::eDistances), nCentroids * _d * sizeof(float), nullptr, 0);
      glNamedBufferStorage(_buffersMedoids(BufferMedoidsType::eIndices), nCentroids * sizeof(uint), nullptr, 0);
      util::KNN knn(dataPtr, _buffersMedoids(BufferMedoidsType::eDistances), _buffersMedoids(BufferMedoidsType::eIndices), _n, 1, _d);
      knn.comp(kMeans.centroids.data(), nCentroids);
      std::vector<uint> indices(nCentroids);
      glGetNamedBufferSubData(_buffersMedoids(BufferMedoidsType::eIndices), 0, nCentroids * sizeof(uint), indices.data());
      glNamedBufferStorage(_bufferResults, nCentroids * _d * sizeof(float), nullptr, GL_DYNAMIC_STORAGE_BIT);
      for (uint i = 0; i < nCentroids; ++i) {
        glCopyNamedBufferSubData(_buffersMedoids(BufferMedoidsType::eDataset), _bufferResults, indices[i] * _d * sizeof(float), i * _d * sizeof(float), _d * sizeof(float));
      }
      glDeleteBuffers(_buffersMedoids.size(), _buffersMedoids.data());
    } else {
      glNamedBufferStorage(_bufferResults, nCentroids * _d * sizeof(float), kMeans.centroids.data(), 0);
    }
    
    // Unmap interop buffers
    for (auto& buffer : _interopBuffers) {
      buffer.unmap();
    }

  }
} // dh::util