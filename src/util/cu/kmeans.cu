/*
 * MIT License
 *
 * Copyright (c) 2021 Mark van de Ruit (Delft University of Technology)
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <faiss/Clustering.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <glad/glad.h>
#include "dh/util/cu/KMeans.cuh"
#include "dh/util/cu/error.cuh"
#include "dh/util/gl/error.hpp"
#include "dh/util/io.hpp"

namespace dh::util {

  KMeans::KMeans() 
  : _isInit(false), _n(0), _d(0), _dataPtr(nullptr) {
    // ...
  }

  KMeans::KMeans(const float* dataPtr, uint n, uint d)
  : _isInit(false), _n(n), _d(d), _dataPtr(dataPtr) {

    _isInit = true;
  }

  // KMeans::KMeans(GLuint datasetBuffer, uint n, uint d, uint levels)
  // : _isInit(false), _n(n), _d(d), _levels(levels), _dataPtr(nullptr) {
    
  //   // Set up OpenGL-CUDA interoperability
  //   _interopBuffers(BufferType::eDataset) = CUGLInteropBuffer(datasetBuffer, CUGLInteropType::eNone);
  //   // _interopBuffers(BufferType::eCentroids) = CUGLInteropBuffer(centroidsBuffer, CUGLInteropType::eNone);

  //   _isInit = true;
  // }

  KMeans::~KMeans() {
    if (_isInit) {
      glDeleteBuffers(1, &_bufferCentroids);
    }
  }

  KMeans::KMeans(KMeans&& other) noexcept {
    swap(*this, other);
  }

  KMeans& KMeans::operator=(KMeans&& other) noexcept {
    swap(*this, other);
    return *this;
  }

  void KMeans::comp(uint nCentroids, bool spherical) {
    // Map interop buffers for access on CUDA side
    // _interopBuffers(BufferType::eCentroids).map();

    const float* dataPtr;
    if(_dataPtr) { dataPtr = _dataPtr; }
    else {
      _interopBuffers(BufferType::eDataset).map();
      dataPtr = (float*) _interopBuffers(BufferType::eDataset).cuHandle();
    }

    // Use a single GPU device. For now, just grab device 0 and pray
    faiss::gpu::StandardGpuResources faissResources;
    faiss::gpu::GpuIndexFlatConfig faissConfig;
    faissConfig.device = 0;
    faissConfig.useFloat16 = false;
    faiss::gpu::GpuIndexFlatL2 faissIndex(&faissResources, _d, faissConfig);

    faiss::ClusteringParameters faissClusteringParams;
    faissClusteringParams.niter = 30;
    faissClusteringParams.verbose = false;
    faissClusteringParams.seed = 0;
    faiss::Clustering kMeans(_d, nCentroids, faissClusteringParams);

    kMeans.train(_n, dataPtr, faissIndex);

    glCreateBuffers(1, &_bufferCentroids);
    glNamedBufferStorage(_bufferCentroids, nCentroids * _d * sizeof(float), kMeans.centroids.data(), 0);
    
    // Unmap interop buffers
    for (auto& buffer : _interopBuffers) {
      buffer.unmap();
    }

  }
} // dh::util